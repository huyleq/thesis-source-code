#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <omp.h>
#include <cstdio>

#include "myio.h"
#include "mylib.h"
#include "init.h"
#include "wave.h"
#include "HessianOp.h"

using namespace std;

int main(int argc,char **argv){
 myio_init(argc,argv);

int nnx,nnz,nx,nz,nt,npad;
float ox,oz,ot,dx,dz,dt;
init2d(nnx,nnz,nx,nz,nt,dx,dz,dt,ox,oz,ot,npad);
int nnxz=nnx*nnz;

 float *wavelet=new float[nt];
 read("wavelet",wavelet,nt);
 
 int ns;
 from_header("souloc","n2",ns);
 float *souloc=new float[ns*4]();
 read("souloc",souloc,ns*4);
 int *sloc=new int[ns*4]();
 #pragma omp parallel for num_threads(16)
 for(int is=0;is<ns;is++){
  sloc[is*4+0]=(souloc[is*4+0]-ox)/dx+0.5+npad;
  sloc[is*4+1]=(souloc[is*4+1]-oz)/dz+0.5+npad;
  sloc[is*4+2]=souloc[is*4+2];
  sloc[is*4+3]=souloc[is*4+3];
 }

 int nr;
 from_header("recloc","n2",nr);
 float *recloc=new float[nr*2]();
 read("recloc",recloc,nr*2);
 int *rloc=new int[nr*2]();
 #pragma omp parallel for num_threads(16)
 for(int ir=0;ir<nr;ir++){
  rloc[ir*2+0]=(recloc[ir*2+0]-ox)/dx+0.5+npad;
  rloc[ir*2+1]=(recloc[ir*2+1]-oz)/dz+0.5+npad;
 }

 float rate;
 get_param("rate",rate);
 int ratio=rate/dt+0.5;
 int ntNeg=std::round(abs(ot)/dt);
 int nnt_data=(nt-ntNeg-1)/ratio+1;
 float *data=new float[nr*nnt_data]();
 read("data",data,nr*nnt_data);

 float *taper=new float[nnxz]();
 init_abc(taper,nx,nz,npad);
 
 float wbottom; get_param("wbottom",wbottom);

 int padded; get_param("padded",padded);
 
 float *m=new float[nnxz]();

 string parameter=get_s("parameter");
 string hesstype=get_s("hesstype");

 if(parameter.compare("vepsdel")==0){
  fprintf(stderr,"parameter v eps del\n");
  float *vepsdel=new float[3*nnxz]();
  float *v=vepsdel,*eps=vepsdel+nnxz,*del=vepsdel+2*nnxz;
  
  float *dvepsdel=new float[3*nnxz]();
  float *dv=dvepsdel,*deps=dvepsdel+nnxz,*ddel=dvepsdel+2*nnxz;
  
  float *gvepsdel=new float[3*nnxz]();
  float *gv=gvepsdel,*geps=gvepsdel+nnxz,*gdel=gvepsdel+2*nnxz;
  
  if(padded==0){
   init_model("v",v,nx,nz,npad); 
   init_model("eps",eps,nx,nz,npad); 
   init_model("del",del,nx,nz,npad); 
   init_model("dv",dv,nx,nz,npad); 
   init_model("deps",deps,nx,nz,npad); 
   init_model("ddel",ddel,nx,nz,npad); 
  } 
  else{ 
   read("v",v,nnxz);
   read("eps",eps,nnxz);
   read("del",del,nnxz);
   read("dv",dv,nnxz);
   read("deps",deps,nnxz);
   read("ddel",ddel,nnxz);
  }
 
  float v0,eps0,del0;
  get_param("v0",v0,"eps0",eps0,"del0",del0);
 
  scale(v,v,1./v0,nnxz);
  scale(eps,eps,1./eps0,nnxz);
  scale(del,del,1./del0,nnxz);
  
  scale(dv,dv,1./v0,nnxz);
  scale(deps,deps,1./eps0,nnxz);
  scale(ddel,ddel,1./del0,nnxz);
  
//  if(hesstype.compare("full")==0) hessianVEpsDel(gv,geps,gdel,data,v,eps,del,dv,deps,ddel,v0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
//  else if(hesstype.compare("GN")==0) GNhessianVEpsDel(gv,geps,gdel,data,v,eps,del,dv,deps,ddel,v0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
  if(hesstype.compare("full")==0){
   HessianOpVEpsDel H(data,v,eps,del,v0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
   H.forward(false,dvepsdel,gvepsdel);
  }
  else if(hesstype.compare("GN")==0){
   GNHessianOpVEpsDel H(data,v,eps,del,v0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
   H.forward(false,dvepsdel,gvepsdel);
  }
  else fprintf(stderr,"please specify hessian type by hesstype=full or hesstype=GN\n");
 
  write("gv",gv,nnxz);
  to_header("gv","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gv","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("geps",geps,nnxz);
  to_header("geps","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("geps","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("gdel",gdel,nnxz);
  to_header("gdel","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gdel","n2",nnz,"o2",-dz*npad,"d2",dz);
 
  delete []vepsdel;
  delete []dvepsdel;
  delete []gvepsdel;
 }
 else if(parameter.compare("vhepsdel")==0){
  fprintf(stderr,"parameter vh eps del\n");
  float *vh=new float[nnxz]();
  float *eps=new float[nnxz]();
  float *del=new float[nnxz]();
  
  float *dvh=new float[nnxz]();
  float *deps=new float[nnxz]();
  float *ddel=new float[nnxz]();
  
  if(padded==0){
   init_model("vh",vh,nx,nz,npad); 
   init_model("eps",eps,nx,nz,npad); 
   init_model("del",del,nx,nz,npad); 
   init_model("dvh",dvh,nx,nz,npad); 
   init_model("deps",deps,nx,nz,npad); 
   init_model("ddel",ddel,nx,nz,npad); 
  } 
  else{ 
   read("vh",vh,nnxz);
   read("eps",eps,nnxz);
   read("del",del,nnxz);
   read("dvh",dvh,nnxz);
   read("deps",deps,nnxz);
   read("ddel",ddel,nnxz);
  }
 
  float vh0,eps0,del0;
  get_param("vh0",vh0,"eps0",eps0,"del0",del0);
 
  scale(vh,vh,1./vh0,nnxz);
  scale(eps,eps,1./eps0,nnxz);
  scale(del,del,1./del0,nnxz);
  
  scale(dvh,dvh,1./vh0,nnxz);
  scale(deps,deps,1./eps0,nnxz);
  scale(ddel,ddel,1./del0,nnxz);
  
  float *gvh=new float[nnxz]();
  float *geps=new float[nnxz]();
  float *gdel=new float[nnxz]();
  
  if(hesstype.compare("full")==0) hessianVhEpsDel(gvh,geps,gdel,data,vh,eps,del,dvh,deps,ddel,vh0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
  else if(hesstype.compare("GN")==0) GNhessianVhEpsDel(gvh,geps,gdel,data,vh,eps,del,dvh,deps,ddel,vh0,eps0,del0,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
  else fprintf(stderr,"please specify hessian type by hesstype=full or hesstype=GN\n");
 
  write("gvh",gvh,nnxz);
  to_header("gvh","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gvh","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("geps",geps,nnxz);
  to_header("geps","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("geps","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("gdel",gdel,nnxz);
  to_header("gdel","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gdel","n2",nnz,"o2",-dz*npad,"d2",dz);
 
  delete []vh; delete []eps; delete []del;
  delete []dvh; delete []deps; delete []ddel;
  delete []gvh; delete []geps; delete []gdel;
 }
 else if(parameter.compare("cij")==0){
  fprintf(stderr,"parameter c11 c13 c33\n");
  float *c11c13c33=new float[3*nnxz]();
  float *c11=c11c13c33,*c13=c11c13c33+nnxz,*c33=c11c13c33+2*nnxz;
  float *dc11c13c33=new float[3*nnxz]();
  float *dc11=dc11c13c33,*dc13=dc11c13c33+nnxz,*dc33=dc11c13c33+2*nnxz;
 
  if(padded==0){
   init_model("c11",c11,nx,nz,npad); 
   init_model("c13",c13,nx,nz,npad); 
   init_model("c33",c33,nx,nz,npad); 
   init_model("dc11",dc11,nx,nz,npad); 
   init_model("dc13",dc13,nx,nz,npad); 
   init_model("dc33",dc33,nx,nz,npad); 
  } 
  else{ 
   read("c11",c11,nnxz);
   read("c13",c13,nnxz);
   read("c33",c33,nnxz);
   read("dc11",dc11,nnxz);
   read("dc13",dc13,nnxz);
   read("dc33",dc33,nnxz);
  }
  
  float c110,c130,c330;
  get_param("c110",c110,"c130",c130,"c330",c330);
 
  scale(c11,c11,1./c110,nnxz);
  scale(c13,c13,1./c130,nnxz);
  scale(c33,c33,1./c330,nnxz);

  scale(dc11,dc11,1./c110,nnxz);
  scale(dc13,dc13,1./c130,nnxz);
  scale(dc33,dc33,1./c330,nnxz);
 
  float *gc11c13c33=new float[3*nnxz]();
  float *gc11=gc11c13c33,*gc13=gc11c13c33+nnxz,*gc33=gc11c13c33+2*nnxz;
 
  if(hesstype.compare("full")==0) hessianCij(gc11,gc13,gc33,data,c11,c13,c33,dc11,dc13,dc33,c110,c130,c330,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
  else if(hesstype.compare("GN")==0) GNhessianCij(gc11,gc13,gc33,data,c11,c13,c33,dc11,dc13,dc33,c110,c130,c330,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot,wbottom,m);
  else fprintf(stderr,"please specify hessian type by hesstype=full or hesstype=GN\n");
 
  write("gc11",gc11,nnxz);
  to_header("gc11","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gc11","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("gc13",gc13,nnxz);
  to_header("gc13","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gc13","n2",nnz,"o2",-dz*npad,"d2",dz);
  write("gc33",gc33,nnxz);
  to_header("gc33","n1",nnx,"o1",-dx*npad,"d1",dx);
  to_header("gc33","n2",nnz,"o2",-dz*npad,"d2",dz);
 
  delete []c11c13c33;
  delete []dc11c13c33;
  delete []gc11c13c33;
 }
 else{
     fprintf(stderr,"please specify parameterization by parameter=something in commandline where something is one of vepsdel, vhepsdel, vnetadel,vhepseta, vvhdel, vnvhdel, cij, or vvnvh\n");
 }
 delete []wavelet;delete []data;delete []sloc;delete []rloc;delete []taper;
 delete []m;
 delete []souloc;delete []recloc;

 myio_close();
 return 0;
}
