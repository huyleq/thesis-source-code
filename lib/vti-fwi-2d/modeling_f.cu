#include "hip/hip_runtime.h"
#include <omp.h>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include "mylib.h"
#include "init.h"
#include "wave.h"
#include "kernels.h"

void modeling_f(float *wavefield,const float *c11,const float *c13,const float *c33,const float *wavelet,int slocxz,const float *taper,int nx,int nz,int nt,int npad,float dx,float dz,float dt,float rate,float ot){
 fprintf(stderr,"Modeling\n");

 int ratio=std::round(rate/dt);
 int ntNeg=std::round(abs(ot)/dt);
 int nnx=nx+2*npad,nnz=nz+2*npad;
 int nnxz=nnx*nnz;
 float dx2=dx*dx,dz2=dz*dz,dt2=dt*dt;
 
 //hipSetDevice(5);

 float *d_wavefieldSlice; 
 hipMalloc(&d_wavefieldSlice,nnxz*sizeof(float));
 hipMemset(d_wavefieldSlice,0,nnxz*sizeof(float));
 
 float *d_c11; hipMalloc(&d_c11,nnxz*sizeof(float));
 float *d_c13; hipMalloc(&d_c13,nnxz*sizeof(float));
 float *d_c33; hipMalloc(&d_c33,nnxz*sizeof(float));
 hipMemcpy(d_c11,c11,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_c13,c13,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_c33,c33,nnxz*sizeof(float),hipMemcpyHostToDevice);
 
 float *d_taper; hipMalloc(&d_taper,nnxz*sizeof(float));
 hipMemcpy(d_taper,taper,nnxz*sizeof(float),hipMemcpyHostToDevice);

 float *p0; hipMalloc(&p0,nnxz*sizeof(float)); 
 float *p1; hipMalloc(&p1,nnxz*sizeof(float)); 
 float *q0; hipMalloc(&q0,nnxz*sizeof(float)); 
 float *q1; hipMalloc(&q1,nnxz*sizeof(float)); 

 dim3 block(BLOCK_DIM_X,BLOCK_DIM_Y);
 dim3 grid((nnx-2*RADIUS+BLOCK_DIM_X-1)/BLOCK_DIM_X,(nnz-2*RADIUS+BLOCK_DIM_Y-1)/BLOCK_DIM_Y);

 hipMemset(p0,0,nnxz*sizeof(float));
 hipMemset(q0,0,nnxz*sizeof(float));
 hipMemset(p1,0,nnxz*sizeof(float));
 hipMemset(q1,0,nnxz*sizeof(float));
  
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 injectSource<<<1,1>>>(p1,q1,dt2*wavelet[0],slocxz);

 abc<<<grid,block>>>(p1,q1,d_taper,nnx,nnz);
 
 if(ratio==1 && ot==0.f){
  recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p1,q1,nnx,nnz);
  hipMemcpy(wavefield+nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(wavefield+nnxz,p1,nnxz*sizeof(float),hipMemcpyDeviceToHost);
 }
 
 for(int it=2;it<nt;++it){
  float t=it*dt+ot;

  forwardCD<<<grid,block>>>(p0,q0,p1,q1,d_c11,d_c13,d_c33,dx2,dz2,dt2,nnx,nnz);

  injectSource<<<1,1>>>(p0,q0,wavelet[it-1],slocxz);

  abc<<<grid,block>>>(p1,q1,p0,q0,d_taper,nnx,nnz);
  
  if(t>=0.f && (it-ntNeg)%ratio==0){
   recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p0,q0,nnx,nnz);
   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,p0,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  }

  float *pt=p0; 
  p0=p1;
  p1=pt;
  pt=q0;
  q0=q1;
  q1=pt;
 }

 hipFree(d_wavefieldSlice);
 hipFree(d_c11);hipFree(d_c13);hipFree(d_c33);
 hipFree(d_taper);
 hipFree(p0);hipFree(p1);hipFree(q0);hipFree(q1);
 
 e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 return;
}

void modelingR_f(float *wavefield,const float *r11,const float *r13,const float *r33,const float *wavelet,int slocxz,const float *taper,int nx,int nz,int nt,int npad,float dx,float dz,float dt,float rate,float ot){
 fprintf(stderr,"Modeling\n");

 int ratio=std::round(rate/dt);
 int ntNeg=std::round(abs(ot)/dt);
 int nnx=nx+2*npad,nnz=nz+2*npad;
 int nnxz=nnx*nnz;
 float dx2=dx*dx,dz2=dz*dz,dt2=dt*dt;
 
 hipSetDevice(5);

 float *d_wavefieldSlice; 
 hipMalloc(&d_wavefieldSlice,nnxz*sizeof(float));
 hipMemset(d_wavefieldSlice,0,nnxz*sizeof(float));
 
 float *d_r11; hipMalloc(&d_r11,nnxz*sizeof(float));
 float *d_r13; hipMalloc(&d_r13,nnxz*sizeof(float));
 float *d_r33; hipMalloc(&d_r33,nnxz*sizeof(float));
 hipMemcpy(d_r11,r11,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_r13,r13,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_r33,r33,nnxz*sizeof(float),hipMemcpyHostToDevice);
 
 float *d_taper; hipMalloc(&d_taper,nnxz*sizeof(float));
 hipMemcpy(d_taper,taper,nnxz*sizeof(float),hipMemcpyHostToDevice);

 float *p0; hipMalloc(&p0,nnxz*sizeof(float)); 
 float *p1; hipMalloc(&p1,nnxz*sizeof(float)); 
 float *q0; hipMalloc(&q0,nnxz*sizeof(float)); 
 float *q1; hipMalloc(&q1,nnxz*sizeof(float)); 

 dim3 block(BLOCK_DIM_X,BLOCK_DIM_Y);
 dim3 grid((nnx-2*RADIUS+BLOCK_DIM_X-1)/BLOCK_DIM_X,(nnz-2*RADIUS+BLOCK_DIM_Y-1)/BLOCK_DIM_Y);

 hipMemset(p0,0,nnxz*sizeof(float));
 hipMemset(q0,0,nnxz*sizeof(float));
 hipMemset(p1,0,nnxz*sizeof(float));
 hipMemset(q1,0,nnxz*sizeof(float));
  
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 injectSource<<<1,1>>>(p1,q1,dt2*wavelet[0],slocxz);

 abc<<<grid,block>>>(p1,q1,d_taper,nnx,nnz);
 
 if(ratio==1 && ot==0.f){
  recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p1,q1,nnx,nnz);
  hipMemcpy(wavefield+nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(wavefield+nnxz,p1,nnxz*sizeof(float),hipMemcpyDeviceToHost);
 }
 
 for(int it=2;it<nt;++it){
  float t=it*dt+ot;

  forwardRDR<<<grid,block>>>(p0,q0,p1,q1,d_r11,d_r13,d_r33,dx2,dz2,dt2,nnx,nnz);

  injectSource<<<1,1>>>(p0,q0,wavelet[it-1],slocxz);

  abc<<<grid,block>>>(p1,q1,p0,q0,d_taper,nnx,nnz);
  
  if(t>=0.f && (it-ntNeg)%ratio==0){
   recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p0,q0,nnx,nnz);
   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,p0,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  }

  float *pt=p0; 
  p0=p1;
  p1=pt;
  pt=q0;
  q0=q1;
  q1=pt;
 }

 hipFree(d_wavefieldSlice);
 hipFree(d_r11);hipFree(d_r13);hipFree(d_r33);
 hipFree(d_taper);
 hipFree(p0);hipFree(p1);hipFree(q0);hipFree(q1);
 
 e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 return;
}

void modelingABCD_f(float *wavefield,const float *a1,const float *b1c1,const float *d1,const float *a2,const float *b2c2,const float *d2,const float *wavelet,int slocxz,const float *taper,int nx,int nz,int nt,int npad,float dx,float dz,float dt,float rate,float ot){
 fprintf(stderr,"Modeling\n");

 int ratio=std::round(rate/dt);
 int ntNeg=std::round(abs(ot)/dt);
 int nnx=nx+2*npad,nnz=nz+2*npad;
 int nnxz=nnx*nnz;
 float dx2=dx*dx,dz2=dz*dz,dt2=dt*dt;
 
 hipSetDevice(5);

 float *d_wavefieldSlice; 
 hipMalloc(&d_wavefieldSlice,nnxz*sizeof(float));
 hipMemset(d_wavefieldSlice,0,nnxz*sizeof(float));
 
 float *d_a1; hipMalloc(&d_a1,nnxz*sizeof(float));
 float *d_b1c1; hipMalloc(&d_b1c1,nnxz*sizeof(float));
 float *d_d1; hipMalloc(&d_d1,nnxz*sizeof(float));
 hipMemcpy(d_a1,a1,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_b1c1,b1c1,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_d1,d1,nnxz*sizeof(float),hipMemcpyHostToDevice);
 
 float *d_a2; hipMalloc(&d_a2,nnxz*sizeof(float));
 float *d_b2c2; hipMalloc(&d_b2c2,nnxz*sizeof(float));
 float *d_d2; hipMalloc(&d_d2,nnxz*sizeof(float));
 hipMemcpy(d_a2,a2,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_b2c2,b2c2,nnxz*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_d2,d2,nnxz*sizeof(float),hipMemcpyHostToDevice);
 
 float *d_taper; hipMalloc(&d_taper,nnxz*sizeof(float));
 hipMemcpy(d_taper,taper,nnxz*sizeof(float),hipMemcpyHostToDevice);

 float *p0; hipMalloc(&p0,nnxz*sizeof(float)); 
 float *p1; hipMalloc(&p1,nnxz*sizeof(float)); 
 float *q0; hipMalloc(&q0,nnxz*sizeof(float)); 
 float *q1; hipMalloc(&q1,nnxz*sizeof(float)); 

 dim3 block(BLOCK_DIM_X,BLOCK_DIM_Y);
 dim3 grid((nnx-2*RADIUS+BLOCK_DIM_X-1)/BLOCK_DIM_X,(nnz-2*RADIUS+BLOCK_DIM_Y-1)/BLOCK_DIM_Y);

 hipMemset(p0,0,nnxz*sizeof(float));
 hipMemset(q0,0,nnxz*sizeof(float));
 hipMemset(p1,0,nnxz*sizeof(float));
 hipMemset(q1,0,nnxz*sizeof(float));
  
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 injectSource<<<1,1>>>(p1,q1,dt2*wavelet[0],slocxz);

 abc<<<grid,block>>>(p1,q1,d_taper,nnx,nnz);
 
 if(ratio==1 && ot==0.f){
  recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p1,q1,nnx,nnz);
  hipMemcpy(wavefield+nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(wavefield+nnxz,p1,nnxz*sizeof(float),hipMemcpyDeviceToHost);
 }
 
 for(int it=2;it<nt;++it){
  float t=it*dt+ot;

  forwardABCD<<<grid,block>>>(p0,q0,p1,q1,d_a1,d_b1c1,d_d1,d_a2,d_b2c2,d_d2,dx2,dz2,dt2,nnx,nnz);

  injectSource<<<1,1>>>(p0,q0,wavelet[it-1],slocxz);

  abc<<<grid,block>>>(p1,q1,p0,q0,d_taper,nnx,nnz);
  
  if(t>=0.f && (it-ntNeg)%ratio==0){
   recordWavefieldSlice<<<grid,block>>>(d_wavefieldSlice,p0,q0,nnx,nnz);
//   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,d_wavefieldSlice,nnxz*sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(wavefield+((it-ntNeg)/ratio)*nnxz,q0,nnxz*sizeof(float),hipMemcpyDeviceToHost);
  }

  float *pt=p0; 
  p0=p1;
  p1=pt;
  pt=q0;
  q0=q1;
  q1=pt;
 }

 hipFree(d_wavefieldSlice);
 hipFree(d_a1);hipFree(d_b1c1);hipFree(d_d1);
 hipFree(d_a2);hipFree(d_b2c2);hipFree(d_d2);
 hipFree(d_taper);
 hipFree(p0);hipFree(p1);hipFree(q0);hipFree(q1);
 
 e=hipGetLastError();
 if(e!=hipSuccess) fprintf(stderr,"error %s\n",hipGetErrorString(e));

 return;
}

