#include "hip/hip_runtime.h"
#include <cmath>
#include <omp.h>
#include <cstdio>
#include "myio.h"
#include "mylib.h"
#include "init.h"
#include "wave.h"
#include "conversions.h"

using namespace std;

int main(int argc,char **argv){
 myio_init(argc,argv);

int nnx,nnz,nx,nz,nt,npad;
 float ox,oz,ot,dx,dz,dt;
 init2d(nnx,nnz,nx,nz,nt,dx,dz,dt,ox,oz,ot,npad);

 float *wavelet=new float[nt];
 read("wavelet",wavelet,nt);
 
 float *c11=new float[nnx*nnz]();
 float *c13=new float[nnx*nnz]();
 float *c33=new float[nnx*nnz]();

 string parameter=get_s("parameter");

 if(parameter.compare("vepsdel")==0){
  fprintf(stderr,"parameter v eps del\n");

  float *v=new float[nnx*nnz]();
  init_model("v",v,nx,nz,npad); 
  float *eps=new float[nnx*nnz]();
  init_model("eps",eps,nx,nz,npad); 
  float *del=new float[nnx*nnz]();
  init_model("del",del,nx,nz,npad); 
 
  VEpsDel2Cij(c11,c13,c33,v,eps,del,1.,1.,1.,nnx*nnz);
  delete []v;delete []eps;delete []del;
 }
 else if(parameter.compare("vhepsdel")==0){
  fprintf(stderr,"parameter vh eps del\n");

  float *vh=new float[nnx*nnz]();
  init_model("vh",vh,nx,nz,npad); 
  float *eps=new float[nnx*nnz]();
  init_model("eps",eps,nx,nz,npad); 
  float *del=new float[nnx*nnz]();
  init_model("del",del,nx,nz,npad); 
 
  VhEpsDel2Cij(c11,c13,c33,vh,eps,del,1.,1.,1.,nnx*nnz);
  delete []vh;delete []eps;delete []del;
 }
 else if(parameter.compare("vnetadel")==0){
  fprintf(stderr,"parameter vn eta del\n");

  float *vn=new float[nnx*nnz]();
  init_model("vn",vn,nx,nz,npad); 
  float *eta=new float[nnx*nnz]();
  init_model("eta",eta,nx,nz,npad); 
  float *del=new float[nnx*nnz]();
  init_model("del",del,nx,nz,npad); 
 
  VnEtaDel2Cij(c11,c13,c33,vn,eta,del,1.,1.,1.,nnx*nnz);
  delete []vn;delete []eta;delete []del;
 }
 else if(parameter.compare("vhepseta")==0){
  fprintf(stderr,"parameter vh eps eta\n");

  float *vh=new float[nnx*nnz]();
  init_model("vh",vh,nx,nz,npad); 
  float *eps=new float[nnx*nnz]();
  init_model("eps",eps,nx,nz,npad); 
  float *eta=new float[nnx*nnz]();
  init_model("eta",eta,nx,nz,npad); 
 
  VhEpsEta2Cij(c11,c13,c33,vh,eps,eta,1.,1.,1.,nnx*nnz);
  delete []vh;delete []eta;delete []eps;
 }
 else if(parameter.compare("vvhdel")==0){
  fprintf(stderr,"parameter v vh del\n");

  float *v=new float[nnx*nnz]();
  init_model("v",v,nx,nz,npad); 
  float *vh=new float[nnx*nnz]();
  init_model("vh",vh,nx,nz,npad); 
  float *del=new float[nnx*nnz]();
  init_model("del",del,nx,nz,npad); 
 
  VVhDel2Cij(c11,c13,c33,v,vh,del,1.,1.,1.,nnx*nnz);
  delete []v;delete []vh;delete []del;
 }
 else if(parameter.compare("vnvhdel")==0){
  fprintf(stderr,"parameter vn vh del\n");

  float *vn=new float[nnx*nnz]();
  init_model("vn",vn,nx,nz,npad); 
  float *vh=new float[nnx*nnz]();
  init_model("vh",vh,nx,nz,npad); 
  float *del=new float[nnx*nnz]();
  init_model("del",del,nx,nz,npad); 
 
  VnVhDel2Cij(c11,c13,c33,vn,vh,del,1.,1.,1.,nnx*nnz);
  delete []vn;delete []vh;delete []del;
 }
 else if(parameter.compare("vvnvh")==0){
  fprintf(stderr,"parameter v vn vh\n");

  float *v=new float[nnx*nnz]();
  init_model("v",v,nx,nz,npad); 
  float *vn=new float[nnx*nnz]();
  init_model("vn",vn,nx,nz,npad); 
  float *vh=new float[nnx*nnz]();
  init_model("vh",vh,nx,nz,npad); 
 
  VVnVh2Cij(c11,c13,c33,v,vn,vh,1.,1.,1.,nnx*nnz);
  delete []v;delete []vh;delete []vn;
 }
 else if(parameter.compare("cij")==0){
  init_model("c11",c11,nx,nz,npad); 
  init_model("c13",c13,nx,nz,npad);
  init_model("c33",c33,nx,nz,npad);
 }
 else{
  fprintf(stderr,"please specify parameterization by parameter=something in commandline where something is one of vepsdel, vhepsdel, vnetadel,vhepseta, vvhdel, vnvhdel, cij, or vvnvh\n");
 }
 
 int ns;
 from_header("souloc","n2",ns);
 float *souloc=new float[ns*4]();
 read("souloc",souloc,ns*4);
 int *sloc=new int[ns*4]();
 #pragma omp parellel for
 for(int is=0;is<ns;is++){
  sloc[is*4+0]=(souloc[is*4+0]-ox)/dx+0.5+npad;
  sloc[is*4+1]=(souloc[is*4+1]-oz)/dz+0.5+npad;
  sloc[is*4+2]=souloc[is*4+2];
  sloc[is*4+3]=souloc[is*4+3];
 }

 int nr;
 from_header("recloc","n2",nr);
 float *recloc=new float[nr*2]();
 read("recloc",recloc,nr*2);
 int *rloc=new int[nr*2]();
 #pragma omp parellel for
 for(int ir=0;ir<nr;ir++){
  rloc[ir*2+0]=(recloc[ir*2+0]-ox)/dx+0.5+npad;
  rloc[ir*2+1]=(recloc[ir*2+1]-oz)/dz+0.5+npad;
 }

 float rate;
 get_param("rate",rate);
 int ratio=rate/dt+0.5;
 int ntNeg=std::round(abs(ot)/dt);
 int nnt_data=(nt-ntNeg-1)/ratio+1;
 float *data=new float[nr*nnt_data]();
 read("data",data,nr*nnt_data);
 
 float *taper=new float[nnx*nnz]();
 init_abc(taper,nx,nz,npad);
 
 float *image=new float[nnx*nnz]();
 float *image1=new float[nnx*nnz]();
 
 rtm_f(image,data,c11,c13,c33,wavelet,sloc,ns,rloc,nr,taper,nx,nz,nt,npad,dx,dz,dt,rate,ot);
 
 for(int iz=1;iz<nnz-1;++iz){
  for(int ix=1;ix<nnx-1;++ix){
   image1[ix+iz*nnx]=image[ix-1+iz*nnx]+image[ix+1+iz*nnx]+image[ix+(iz-1)*nnx]+image[ix+(iz+1)*nnx]-4.*image[ix+iz*nnx];
  }
 }

 float wbottom; get_param("wbottom",wbottom);
 int n=npad+wbottom/dz+1;
 memset(image1,0,n*nnx*sizeof(float)); 

 write("image",image1,nnx*nnz);
 to_header("image","n1",nnx,"o1",-dx*npad,"d1",dx);
 to_header("image","n2",nnz,"o2",-dz*npad,"d2",dz);
 
 delete []wavelet;delete []data;delete []sloc;delete []rloc;delete []taper;
 delete []c11;delete []c13;delete []c33;delete []image;delete []image1;
 delete []souloc;delete []recloc;

 myio_close();
 return 0;
}
