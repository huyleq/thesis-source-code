#include "hip/hip_runtime.h"
#include <omp.h>
#include <cstdio>
#include <cstdlib>
#include "myio.h"
#include "mylib.h"
#include "init.h"
#include "myio.h"
#include "wave.h"
#include "kernels.h"

#include <vector>

void rtm_f(float *image,const float *data,const float *c11,const float *c13,const float *c33,const float *wavelet,const int *sloc,int ns,const int *rloc,int nr,const float *taper,int nx,int nz,int nt,int npad,float dx,float dz,float dt,float rate,float ot){
 fprintf(stderr,"Starting migration...\n");

 int ratio=rate/dt+0.5f;
 int ntNeg=std::round(abs(ot)/dt);
 int nnt=(nt-1)/ratio+1;
 int nnt_data=(nt-ntNeg-1)/ratio+1;
 int nnx=nx+2*npad,nnz=nz+2*npad;
 int nnxz=nnx*nnz;
 float dx2=dx*dx,dz2=dz*dz,dt2=dt*dt;
 
 memset(image,0,nnxz*sizeof(float));
 
 std::vector<int> GPUs;
 get_array("gpu",GPUs);
 int nGPUs=GPUs.size();
 fprintf(stderr,"Total # GPUs = %d\n",nGPUs);
 fprintf(stderr,"GPUs used are:\n");
 for(int i=0;i<nGPUs;i++) fprintf(stderr,"%d",GPUs[i]);
 fprintf(stderr,"\n");

 float **sourceWavefield=new float*[nGPUs]();
// float **recWavefield=new float*[nGPUs]();
 int **d_rloc=new int*[nGPUs]();
 float **d_c11=new float*[nGPUs]();
 float **d_c13=new float*[nGPUs]();
 float **d_c33=new float*[nGPUs]();
 float **d_taper=new float*[nGPUs]();
 float **d_sourceWavefieldSlice=new float*[nGPUs]();
 float **p0=new float*[nGPUs]();
 float **q0=new float*[nGPUs]();
 float **p1=new float*[nGPUs]();
 float **q1=new float*[nGPUs]();
 float **d_image=new float*[nGPUs]();
 float **d_sourceWavefieldSlice0=new float*[nGPUs]();
 float **d_sourceWavefieldSlice1=new float*[nGPUs]();
 float **d_data0=new float*[nGPUs]();
 float **d_data1=new float*[nGPUs]();
 float **images=new float*[nGPUs]();
 
 for(int i=0;i<nGPUs;++i){
  hipSetDevice(GPUs[i]);
  
  sourceWavefield[i]=new float[nnxz*nnt]();
//  recWavefield[i]=new float[nnxz*nnt]();
  
  hipMalloc(&d_c11[i],nnxz*sizeof(float));
  hipMalloc(&d_c13[i],nnxz*sizeof(float));
  hipMalloc(&d_c33[i],nnxz*sizeof(float));
  hipMemcpy(d_c11[i],c11,nnxz*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_c13[i],c13,nnxz*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_c33[i],c33,nnxz*sizeof(float),hipMemcpyHostToDevice);
  
  hipMalloc(&d_taper[i],nnxz*sizeof(float));
  hipMemcpy(d_taper[i],taper,nnxz*sizeof(float),hipMemcpyHostToDevice);
 
  hipMalloc(&d_sourceWavefieldSlice[i],nnxz*sizeof(float));
  hipMemset(d_sourceWavefieldSlice[i],0,nnxz*sizeof(float));
 
  hipMalloc(&p0[i],nnxz*sizeof(float)); 
  hipMalloc(&p1[i],nnxz*sizeof(float)); 
  hipMalloc(&q0[i],nnxz*sizeof(float)); 
  hipMalloc(&q1[i],nnxz*sizeof(float)); 
  
  hipMalloc(&d_image[i],nnxz*sizeof(float));
  hipMemset(d_image[i],0,nnxz*sizeof(float));
  
  hipMalloc(&d_sourceWavefieldSlice0[i],nnxz*sizeof(float));
  hipMalloc(&d_sourceWavefieldSlice1[i],nnxz*sizeof(float));
  hipMemset(d_sourceWavefieldSlice0[i],0,nnxz*sizeof(float));
  hipMemset(d_sourceWavefieldSlice1[i],0,nnxz*sizeof(float));
 
  images[i]=new float[nnxz]();
 }

 int npasses=(ns+nGPUs-1)/nGPUs;
 int shotLeft=ns;

 for(int pass=0;pass<npasses;++pass){
  int nGPUsNeed=min(shotLeft,nGPUs);
  fprintf(stderr,"Pass %d, # GPUs = %d\n",pass,nGPUsNeed);
  
  #pragma omp parallel for num_threads(nGPUsNeed)
  for(int i=0;i<nGPUsNeed;++i){
  hipSetDevice(GPUs[i]);

   int is=pass*nGPUs+i;
   int slocxz=sloc[0+is*4]+sloc[1+is*4]*nnx;

   hipMalloc(&d_rloc[i],2*sloc[2+is*4]*sizeof(int));
   hipMemcpy(d_rloc[i],rloc+2*sloc[3+is*4],2*sloc[2+is*4]*sizeof(int),hipMemcpyHostToDevice);

   dim3 block(BLOCK_DIM_X,BLOCK_DIM_Y);
   dim3 grid((nnx-2*RADIUS+BLOCK_DIM_X-1)/BLOCK_DIM_X,(nnz-2*RADIUS+BLOCK_DIM_Y-1)/BLOCK_DIM_Y);

   hipMemset(p0[i],0,nnxz*sizeof(float));
   hipMemset(q0[i],0,nnxz*sizeof(float));
   hipMemset(p1[i],0,nnxz*sizeof(float));
   hipMemset(q1[i],0,nnxz*sizeof(float));
 
   injectDipoleSource<<<1,1>>>(p1[i],q1[i],dt2*wavelet[0],slocxz,nnx);
  
   abc<<<grid,block>>>(p1[i],q1[i],d_taper[i],nnx,nnz);
  
   if(ratio==1){
    recordWavefieldSlice<<<grid,block>>>(d_sourceWavefieldSlice[i],p1[i],q1[i],nnx,nnz);
    hipMemcpy(sourceWavefield[i]+nnxz,d_sourceWavefieldSlice[i],nnxz*sizeof(float),hipMemcpyDeviceToHost);
   }
   
   for(int it=2;it<nt;++it){
    //fprintf(stderr,"Time step it=%d\n",it);
  
    forwardCD<<<grid,block>>>(p0[i],q0[i],p1[i],q1[i],d_c11[i],d_c13[i],d_c33[i],dx2,dz2,dt2,nnx,nnz);
  
    injectDipoleSource<<<1,1>>>(p0[i],q0[i],dt2*wavelet[it-1],slocxz,nnx);
  
    abc<<<grid,block>>>(p1[i],q1[i],p0[i],q0[i],d_taper[i],nnx,nnz);
    
    if(it%ratio==0){
     recordWavefieldSlice<<<grid,block>>>(d_sourceWavefieldSlice[i],p0[i],q0[i],nnx,nnz);
     hipMemcpy(sourceWavefield[i]+(it/ratio)*nnxz,d_sourceWavefieldSlice[i],nnxz*sizeof(float),hipMemcpyDeviceToHost);
    }
  
    float *pt=p0[i]; 
    p0[i]=p1[i];
    p1[i]=pt;
    pt=q0[i];
    q0[i]=q1[i];
    q1[i]=pt;
   }

//   write("sourceWavefield",sourceWavefield[i],nnxz*nnt);
//   to_header("sourceWavefield","n1",nnx,"o1",-dx*npad,"d1",dx);
//   to_header("sourceWavefield","n2",nnz,"o2",-dz*npad,"d2",dz);
//   to_header("sourceWavefield","n3",nnt,"o3",ot,"d3",rate);

//   hipMemset(d_sourceWavefieldSlice[i],0,nnxz*sizeof(float));

   hipMemset(p0[i],0,nnxz*sizeof(float));
   hipMemset(q0[i],0,nnxz*sizeof(float));
   hipMemset(p1[i],0,nnxz*sizeof(float));
   hipMemset(q1[i],0,nnxz*sizeof(float));
  
   hipMalloc(&d_data0[i],sloc[2+is*4]*sizeof(float));
   hipMalloc(&d_data1[i],sloc[2+is*4]*sizeof(float));
   hipMemset(d_data0[i],0,sloc[2+is*4]*sizeof(float));
   hipMemset(d_data1[i],0,sloc[2+is*4]*sizeof(float));
  
   hipMemcpy(d_data0[i],data+(nnt_data-1)*nr+sloc[3+is*4],sloc[2+is*4]*sizeof(float),hipMemcpyHostToDevice);
   injectDipoleData<<<(sloc[2+is*4]+BLOCK_DIM_X-1)/BLOCK_DIM_X,BLOCK_DIM_X>>>(p0[i],q0[i],d_data0[i],d_data1[i],0.f,d_rloc[i],sloc[2+is*4],nnx,dt2);
  
   abc<<<grid,block>>>(p0[i],q0[i],d_taper[i],nnx,nnz);
  
   float f=(nt-2.)/ratio;
   int i1=f,i2=i1+1;
   hipMemcpy(d_sourceWavefieldSlice0[i],sourceWavefield[i]+i1*nnxz,nnxz*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpy(d_sourceWavefieldSlice1[i],sourceWavefield[i]+i2*nnxz,nnxz*sizeof(float),hipMemcpyHostToDevice);
   f=f-i1;
   imagingCrossCor<<<grid,block>>>(d_image[i],p0[i],q0[i],d_sourceWavefieldSlice0[i],d_sourceWavefieldSlice1[i],f,nnx,nnz);
  
   for(int it=nt-3;it>=0;--it){
    //fprintf(stderr,"Time step it=%d\n",it);
    
//    backwardDC<<<grid,block>>>(p1[i],q1[i],p0[i],q0[i],d_c11[i],d_c13[i],d_c33[i],dx2,dz2,dt2,nnx,nnz);
    forwardCD<<<grid,block>>>(p1[i],q1[i],p0[i],q0[i],d_c11[i],d_c13[i],d_c33[i],dx2,dz2,dt2,nnx,nnz);
    
	if(it>=ntNeg){
     f=(it-ntNeg+1.)/ratio;
     i1=f;
     if((it-ntNeg+2)%ratio==0){
      hipMemcpy(d_data1[i],data+i1*nr+sloc[3+is*4],sloc[2+is*4]*sizeof(float),hipMemcpyHostToDevice);
 	 float *pt=d_data0[i];
      d_data0[i]=d_data1[i];
      d_data1[i]=pt;
     }
     f=f-i1;
     injectDipoleData<<<(sloc[2+is*4]+BLOCK_DIM_X-1)/BLOCK_DIM_X,BLOCK_DIM_X>>>(p1[i],q1[i],d_data0[i],d_data1[i],f,d_rloc[i],sloc[2+is*4],nnx,dt2);
	}

    abc<<<grid,block>>>(p1[i],q1[i],p0[i],q0[i],d_taper[i],nnx,nnz);
  
//    if(it%ratio==0){
//     recordWavefieldSlice<<<grid,block>>>(d_sourceWavefieldSlice[i],p0[i],q0[i],nnx,nnz);
//     hipMemcpy(recWavefield[i]+(it/ratio)*nnxz,d_sourceWavefieldSlice[i],nnxz*sizeof(float),hipMemcpyDeviceToHost);
//    }
    
    f=(float)it/ratio;
    i1=f;
    if((it+1)%ratio==0){
     hipMemcpy(d_sourceWavefieldSlice1[i],sourceWavefield[i]+i1*nnxz,nnxz*sizeof(float),hipMemcpyHostToDevice);
     float *pt=d_sourceWavefieldSlice0[i]; 
     d_sourceWavefieldSlice0[i]=d_sourceWavefieldSlice1[i];
     d_sourceWavefieldSlice1[i]=pt;
    }
    f=f-i1;
    imagingCrossCor<<<grid,block>>>(d_image[i],p1[i],q1[i],d_sourceWavefieldSlice0[i],d_sourceWavefieldSlice1[i],f,nnx,nnz);
  
    float *pt=p0[i]; 
    p0[i]=p1[i];
    p1[i]=pt;
    pt=q0[i];
    q0[i]=q1[i];
    q1[i]=pt;
   }
   
//   write("recWavefield",recWavefield[i],nnxz*nnt);
//   to_header("recWavefield","n1",nnx,"o1",-dx*npad,"d1",dx);
//   to_header("recWavefield","n2",nnz,"o2",-dz*npad,"d2",dz);
//   to_header("recWavefield","n3",nnt,"o3",ot,"d3",rate);

   hipFree(d_rloc[i]);hipFree(d_data0[i]);hipFree(d_data1[i]);
  }
  
  shotLeft-=nGPUsNeed;
 }

 #pragma omp parallel for num_threads(nGPUs)
 for(int i=0;i<nGPUs;i++){
   hipSetDevice(GPUs[i]);
   hipMemcpy(images[i],d_image[i],nnxz*sizeof(float),hipMemcpyDeviceToHost);
   hipDeviceSynchronize();
 }
  
  for(int i=0;i<nGPUs;++i){
   #pragma omp parallel for num_threads(16) shared(i)
   for(size_t ixz=0;ixz<nnxz;++ixz){
    image[ixz]+=images[i][ixz];
   }
  }

 for(int i=0;i<nGPUs;++i){
  hipSetDevice(GPUs[i]);
  delete []sourceWavefield[i];
  hipFree(d_c11[i]);hipFree(d_c13[i]);hipFree(d_c33[i]); 
  hipFree(d_taper[i]);
  hipFree(d_sourceWavefieldSlice[i]);
  hipFree(p0[i]);hipFree(p1[i]);hipFree(q0[i]);hipFree(q1[i]);
  hipFree(d_image[i]);
  hipFree(d_sourceWavefieldSlice0[i]);hipFree(d_sourceWavefieldSlice1[i]);
  delete []images[i];
  
  hipError_t e=hipGetLastError();
  if(e!=hipSuccess) fprintf(stderr,"gpu %d error %s\n",GPUs[i],hipGetErrorString(e));
 }
 
 delete []sourceWavefield;
 delete []d_rloc;
 delete []d_c11;delete []d_c13;delete []d_c33;
 delete []d_taper;
 delete []d_sourceWavefieldSlice;
 delete []p0;delete []p1;delete []q0;delete []q1;
 delete []d_image;
 delete []d_sourceWavefieldSlice0;delete []d_sourceWavefieldSlice1;delete []d_data0;delete []d_data1;
 delete []images;

 return;
}
