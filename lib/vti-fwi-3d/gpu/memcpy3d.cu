#include "wave3d.h"

void memcpyCpuToCpu2(float *dest1,float *sou1,float *dest2,float *sou2,size_t nbytes){
 memcpy(dest1,sou1,nbytes);
 memcpy(dest2,sou2,nbytes);
}

void memcpyCpuToCpu3(float *dest1,float *sou1,float *dest2,float *sou2,float *dest3,float *sou3,size_t nbytes){
 memcpy(dest1,sou1,nbytes);
 memcpy(dest2,sou2,nbytes);
 memcpy(dest3,sou3,nbytes);
}

void memcpyCpuToGpu2(float *dest1,float *sou1,float *dest2,float *sou2,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyHostToDevice,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyHostToDevice,*stream);
}

void memcpyCpuToGpu3(float *dest1,float *sou1,float *dest2,float *sou2,float *dest3,float *sou3,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyHostToDevice,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyHostToDevice,*stream);
 hipMemcpyAsync(dest3,sou3,nbytes,hipMemcpyHostToDevice,*stream);
}

void memcpyGpuToCpu2(float *dest1,float *sou1,float *dest2,float *sou2,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyDeviceToHost,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyDeviceToHost,*stream);
}

void memcpyGpuToCpu3(float *dest1,float *sou1,float *dest2,float *sou2,float *dest3,float *sou3,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyDeviceToHost,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyDeviceToHost,*stream);
 hipMemcpyAsync(dest3,sou3,nbytes,hipMemcpyDeviceToHost,*stream);
}

void memcpyGpuToGpu2(float *dest1,float *sou1,float *dest2,float *sou2,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyDefault,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyDefault,*stream);
}

void memcpyGpuToGpu3(float *dest1,float *sou1,float *dest2,float *sou2,float *dest3,float *sou3,size_t nbytes,hipStream_t *stream){
 hipMemcpyAsync(dest1,sou1,nbytes,hipMemcpyDefault,*stream);
 hipMemcpyAsync(dest2,sou2,nbytes,hipMemcpyDefault,*stream);
 hipMemcpyAsync(dest3,sou3,nbytes,hipMemcpyDefault,*stream);
}

