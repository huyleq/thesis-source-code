#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <chrono>

#include "myio.h"
#include "mylib.h"
#include "wave3d.h"
#include "conversions.h"

using namespace std;

int main(int argc,char **argv){
 myio_init(argc,argv);
 
 int nx,ny,nz,npad,nt;
 float ox,oy,oz,ot,dx,dy,dz,dt;
 
 from_header("v","n1",nx,"o1",ox,"d1",dx);
 from_header("v","n2",ny,"o2",oy,"d2",dy);
 from_header("v","n3",nz,"o3",oz,"d3",dz);
 get_param("npad",npad);
 get_param("nt",nt,"ot",ot,"dt",dt);
 
 long long nxy=nx*ny;
 long long nxyz=nxy*nz;
 
 float *wavelet=new float[nt]();
 read("wavelet",wavelet,nt);

 float samplingRate;
 get_param("samplingRate",samplingRate);
 int samplingTimeStep=std::round(samplingRate/dt);
// int nnt=(nt-1)/samplingTimeStep+1;

 float *v=new float[nxyz];
 read("v",v,nxyz);
 float *eps=new float[nxyz]();
 read("eps",eps,nxyz);
 float *del=new float[nxyz]();
 read("del",del,nxyz);

 float *c11=new float[nxyz];
 float *c13=new float[nxyz];
 float *c33=new float[nxyz];
 VEpsDel2Cij(c11,c13,c33,v,eps,del,1.,1.,1.,nxyz);

 float soulocX,soulocY,soulocZ;
 get_param("soulocX",soulocX,"soulocY",soulocY,"soulocZ",soulocZ);

 fprintf(stderr,"modeling\n");

 chrono::high_resolution_clock::time_point start=chrono::high_resolution_clock::now();
 
 modeling3d_f(soulocX,soulocY,soulocZ,wavelet,c11,c13,c33,nx,ny,nz,nt,npad,ox,oy,oz,ot,dx,dy,dz,dt);

 chrono::high_resolution_clock::time_point end=chrono::high_resolution_clock::now();
 chrono::duration<double> time=chrono::duration_cast<chrono::duration<double> >(end-start);
 cout<<"total time "<<time.count()/60.<<" minutes"<<endl;
 
 delete []wavelet;
 delete []v;delete []eps;delete []del;
 delete []c11;delete []c13;delete []c33;

 myio_close();
 return 0;
}

