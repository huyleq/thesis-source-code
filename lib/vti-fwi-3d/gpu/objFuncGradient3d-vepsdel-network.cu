#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>
#include <string>
#include <vector>
#include <algorithm>
#include <thread>

#include <errno.h>
#include <string.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <libssh/libssh.h>
#include <libssh/sftp.h>

#include "sshtunneling.h"

#include "myio.h"
#include "mylib.h"
#include "wave3d.h"
#include "objFuncGradientCij3d_network.h"
#include "conversions.h"
#include "boundary.h"
#include "check.h"
#include "lbfgs.h"

using namespace std;

int main(int argc,char **argv){
 myio_init(argc,argv);

 int nx,ny,nz,npad;
 float ox,oy,oz,dx,dy,dz;
 
 from_header("v","n1",nx,"o1",ox,"d1",dx);
 from_header("v","n2",ny,"o2",oy,"d2",dy);
 from_header("v","n3",nz,"o3",oz,"d3",dz);
 get_param("npad",npad);
 
 long long nxy=nx*ny,nxyz=nxy*nz,nn=3*nxyz;
 float f;
 float *vepsdel=new float[nn]; 
 float *v=vepsdel,*eps=vepsdel+nxyz,*del=vepsdel+2*nxyz;
 float *m=new float[nxyz]();
 float *cij=new float[nn];
 float *c11=cij,*c13=cij+nxyz,*c33=cij+2*nxyz;
 float *mask=new float[nxyz];
 
 float v0=1.,eps0=1.,wbottom=0.;
 get_param("v0",v0,"eps0",eps0,"wbottom",wbottom);
 read("v",v,nxyz);
 scale(v,v,1./v0,nxyz);
 if(!read("eps",eps,nxyz)) memset(eps,0,nxyz*sizeof(float));
 scale(eps,eps,1./eps0,nxyz);
 if(!read("del",del,nxyz)) memset(del,0,nxyz*sizeof(float));
 if(!read("mask",mask,nxyz)) set(mask,1.f,nxyz);

 // remote server part 
 vector<string> server; get_sarray("server",server);
 int nserver=server.size();

 float **fgcij=new float*[nserver]();
 vector<string> workdir; get_sarray("remoteworkdir",workdir);
 vector<string> datapath; get_sarray("remotedatapath",datapath);
 vector<string> script; get_sarray("remotescript",script,";");
 vector<string> command; get_sarray("remotecommand",command,";");
 vector<string> script1,scriptpath,gradpath,outpath,command1;

 ssh_session *my_ssh_session=new ssh_session[nserver]();
 sftp_session *my_sftp_session=new sftp_session[nserver]();
 
 string homedir(getenv("HOME"));
 
 for(int i=0;i<nserver;i++){
     fgcij[i]=new float[nn+1];

     scriptpath.push_back(workdir[i]+"scripts/");
     gradpath.push_back(workdir[i]+"grads/");
     outpath.push_back(workdir[i]+"output/");
     script1.push_back("#!/usr/bin/env tcsh\n\ncd "+workdir[i]+"\n\n"+script[i]+" datapath="+datapath[i]);
     if(command[i].compare("/bin/bash")==0) command1.push_back(command[i]);
     else command1.push_back(workdir[i]+command[i]);
     
//     cout<<"remote workdir "<<workdir[i]<<endl;
//     cout<<"remote datapath "<<datapath[i]<<endl;
//     cout<<"remote scriptpath "<<scriptpath[i]<<endl;
//     cout<<"remote gradpath "<<gradpath[i]<<endl;
//     cout<<"remote outpath "<<outpath[i]<<endl;
//     cout<<"script1 "<<script1[i]<<endl;
//     cout<<"command1 "<<command1[i]<<endl;
    
     // Open session and set options
     my_ssh_session[i]=ssh_new();
     if (my_ssh_session[i] == NULL) exit(-1);
     cout<<"conneting to "<<server[i]<<endl;
     string ipaddr=get_s(homedir+"/.ipaddr",server[i]);
     ssh_options_set(my_ssh_session[i], SSH_OPTIONS_HOST, ipaddr.c_str());
     
     // Connect to server
     int rc = ssh_connect(my_ssh_session[i]);
     if (rc != SSH_OK){
       fprintf(stderr, "Error connecting to %s: %s\n",server[i].c_str(),ssh_get_error(my_ssh_session[i]));
       ssh_free(my_ssh_session[i]);
       exit(-1);
     }
    
     // Authenticate ourselves
     string pass=get_s(homedir+"/.pass",server[i]);
     rc = ssh_userauth_password(my_ssh_session[i], NULL, pass.c_str());
     if (rc != SSH_AUTH_SUCCESS){
       fprintf(stderr, "Error authenticating with password when connectin to %s: %s\n",server[i].c_str(),ssh_get_error(my_ssh_session[i]));
       ssh_disconnect(my_ssh_session[i]);
       ssh_free(my_ssh_session[i]);
       exit(-1);
     }
    
     my_sftp_session[i] = sftp_new(my_ssh_session[i]);
     if (my_sftp_session[i] == NULL) {
       fprintf(stderr, "Error allocating SFTP session: %s\n",ssh_get_error(my_ssh_session[i]));
       return SSH_ERROR;
     }
     rc = sftp_init(my_sftp_session[i]);
     if (rc != SSH_OK) {
       fprintf(stderr, "Error initializing SFTP session: %d.\n",sftp_get_error(my_sftp_session[i]));
       sftp_free(my_sftp_session[i]);
       return rc;
     }
 }

 vector<thread> threads;

 // cluster part 
 float *fgcij_cluster=new float[nn+1]();
 string cluster_command=get_s("command");

 vector<int> shotid;
 vector<int> shotrange; get_array("shotrange",shotrange);
 vector<int> badshot; get_array("badshot",badshot);
 for(int i=shotrange[0];i<shotrange[1];i++){
  if(find(badshot.begin(),badshot.end(),i)==badshot.end()) shotid.push_back(i);
 }

 int max_shot_per_job=1;
 float pct=1.;
 get_param("max_shot_per_job",max_shot_per_job,"pct",pct);
    
 chrono::high_resolution_clock::time_point start=chrono::high_resolution_clock::now();
 
 int icall;
 get_param("icall",icall);

 checkEpsDel(eps,del,eps0,1.,nxyz,m);
 VEpsDel2Cij(c11,c13,c33,v,eps,del,v0,eps0,1.,nxyz);
 
 string cijfile="cij_icall_"+to_string(icall)+".H";

 for(int i=0;i<nserver;i++) threads.push_back(thread(objFuncGradientCij3d_network,fgcij[i],cij,nx,ny,nz,ox,oy,oz,dx,dy,dz,std::ref(cijfile),std::ref(script1[i]),std::ref(scriptpath[i]),std::ref(gradpath[i]),std::ref(outpath[i]),std::ref(datapath[i]),std::ref(command1[i]),icall,std::ref(my_ssh_session[i]),std::ref(my_sftp_session[i])));
 
 writeToHeader(cijfile,cij,nn);
 
 ofstream ofs;
 if(!open_file(ofs,cijfile,ofstream::app)){
     cout<<"cannot open file "<<cijfile<<endl;
 }
 else{
     ofs<<"n1="<<nx<<" o1="<<ox<<" d1="<<dx<<endl;
     ofs<<"n2="<<ny<<" o2="<<oy<<" d2="<<dy<<endl;
     ofs<<"n3="<<nz<<" o3="<<oz<<" d3="<<dz<<endl;
     ofs<<"n4="<<3<<" o4="<<0<<" d4="<<1<<endl;
 }
 close_file(ofs);

 string cluster_command1=cluster_command+" cij="+cijfile;

 if(shotid.size()>0) objFuncGradientCij_cluster(fgcij_cluster,nx,ny,nz,shotid,pct,max_shot_per_job,icall,cluster_command1);
 
 for(int i=0;i<nserver;i++) threads[i].join();

 for(int i=0;i<nserver;i++){
    #pragma omp parallel for
    for(size_t j=0;j<nn+1;j++) fgcij_cluster[j]+=fgcij[i][j];
 }
 
 f=fgcij_cluster[0];
 fprintf(stderr,"objfunc is %10.16f\n",f); 

 float *gvepsdel=new float[nn];
 float *gv=gvepsdel,*geps=gvepsdel+nxyz,*gdel=gvepsdel+2*nxyz;
 float *gcij=fgcij_cluster+1;
 float *gc11=gcij,*gc13=gcij+nxyz,*gc33=gcij+2*nxyz;
 
 zeroBoundary(gc11,nx,ny,nz,npad);
 zeroBoundary(gc13,nx,ny,nz,npad);
 zeroBoundary(gc33,nx,ny,nz,npad);
 
 int nwbottom=(wbottom-oz)/dz+1-npad;
 memset(gc11+npad*nxy,0,nwbottom*nxy*sizeof(float));
 memset(gc13+npad*nxy,0,nwbottom*nxy*sizeof(float));
 memset(gc33+npad*nxy,0,nwbottom*nxy*sizeof(float));
 
 GradCij2GradVEpsDel(gv,geps,gdel,gc11,gc13,gc33,v,eps,del,v0,eps0,1.,nxyz);
 
 multiply(gv,gv,mask,nxyz);
 multiply(geps,geps,mask,nxyz);
 multiply(gdel,gdel,mask,nxyz);

 chrono::high_resolution_clock::time_point end=chrono::high_resolution_clock::now();
 chrono::duration<double> time=chrono::duration_cast<chrono::duration<double> >(end-start);
 cout<<"total time "<<time.count()/60.<<" minutes"<<endl;
 
 float agv=fabs(gv[0]),av=fabs(v[0]);
 float age=fabs(geps[0]),ae=fabs(eps[0]);
 float agd=fabs(gdel[0]),ad=fabs(del[0]);
 for(int i=0;i<nxyz;i++){
     if(fabs(gv[i])>agv) agv=fabs(gv[i]);
     if(fabs(v[i])>av) av=fabs(v[i]);
     if(fabs(geps[i])>age) age=fabs(geps[i]);
     if(fabs(eps[i])>ae) ae=fabs(eps[i]);
     if(fabs(gdel[i])>agd) agd=fabs(gdel[i]);
     if(fabs(del[i])>ad) ad=fabs(del[i]);
 }

 if(ae==0.) get_param("maxeps",ae);
 if(ad==0.) get_param("maxdel",ad);

 v0=sqrt((agd/ad)/(agv/av));
 eps0=sqrt((agd/ad)/(age/ae));

 cout<<"v0 should be "<<v0<<endl;
 cout<<"eps0 should be "<<eps0<<endl;
 cout<<"del0 should be 1"<<endl;

 write("gv",gv,nxyz);
 to_header("gv","n1",nx,"o1",ox,"d1",dx);
 to_header("gv","n2",ny,"o2",oy,"d2",dy);
 to_header("gv","n3",nz,"o3",oz,"d3",dz);

 write("geps",geps,nxyz);
 to_header("geps","n1",nx,"o1",ox,"d1",dx);
 to_header("geps","n2",ny,"o2",oy,"d2",dy);
 to_header("geps","n3",nz,"o3",oz,"d3",dz);

 write("gdel",gdel,nxyz);
 to_header("gdel","n1",nx,"o1",ox,"d1",dx);
 to_header("gdel","n2",ny,"o2",oy,"d2",dy);
 to_header("gdel","n3",nz,"o3",oz,"d3",dz);

 for(int i=0;i<nserver;i++){
     delete []fgcij[i];
     sftp_free(my_sftp_session[i]);
     ssh_disconnect(my_ssh_session[i]);
     ssh_free(my_ssh_session[i]);
 }
 delete []fgcij;
 delete []my_ssh_session;delete []my_sftp_session;
 delete []vepsdel;delete []cij;delete []gvepsdel;delete []fgcij_cluster;delete []mask;
 delete []m;

 myio_close();
 return 0;
}
